#include "hip/hip_runtime.h"
//Copyright (c) 2012, Mikhail Sirotenko <mihail.sirotenko@gmail.com>
//All rights reserved.
//
//Redistribution and use in source and binary forms, with or without
//modification, are permitted provided that the following conditions are met:
//    * Redistributions of source code must retain the above copyright
//      notice, this list of conditions and the following disclaimer.
//    * Redistributions in binary form must reproduce the above copyright
//      notice, this list of conditions and the following disclaimer in the
//      documentation and/or other materials provided with the distribution.
//
//THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
//ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
//WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
//DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
//DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
//(INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
//LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
//ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
//(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
//SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <stdexcept>
#include "../precomp.hpp"

namespace cudacnn
{

//Instantiate
template class PoolingLayer<TensorGPU, float>;
template class PoolingLayer<TensorGPU, double>;

#ifdef HAVE_CUDA

template <class T, int nthreads>
__global__ void SubsampleKernel(const TensorDev3<T> inputs, TensorDev3<T> output)
{
	int sx = inputs.w / output.w;
	int sy = inputs.h / output.h;
	int tx = threadIdx.x%sx;
	int ty = threadIdx.x/sx;
	int x = blockIdx.x%output.w; 
	int y = blockIdx.x/output.w;
	int m = blockIdx.y;

	int tid = threadIdx.x;
	__shared__ T smem[nthreads];
	smem[tid] = 0;

	if(tx < sx && ty < sy){
		smem[tid] = inputs(x*sx + tx, y*sy + ty, m);
	}
	volatile T* vsmem = smem;
	////Reduction
	__syncthreads();
	if(nthreads >= 256){
		if(tid < 128) vsmem[tid] += vsmem[tid+128];
		__syncthreads();
	}
	if(nthreads >= 128){
		if(tid < 64) 
			vsmem[tid] += vsmem[tid+64];
		__syncthreads();
	}

	//All these run in a single warp
	if(nthreads >= 64) if(tid < 32) vsmem[tid] += vsmem[tid + 32];
	if(nthreads >= 32) if(tid < 16) vsmem[tid] += vsmem[tid + 16];
	if(nthreads >= 16) if(tid < 8) vsmem[tid] += vsmem[tid + 8];
	if(nthreads >= 8)  if(tid < 4) vsmem[tid] += vsmem[tid + 4];
	if(nthreads >= 4)  if(tid < 2) vsmem[tid] += vsmem[tid + 2];
	if(nthreads >= 2)  if(tid < 1) vsmem[tid] += vsmem[tid + 1];

	if(tid == 0)
		output(x,y,m) = vsmem[0]/(sx*sy);
}
template <class T, int nthreads>
__global__ void MaxPoolingKernel(const TensorDev3<T> inputs, TensorDev3<T> output)
{
	int sx = inputs.w / output.w;
	int sy = inputs.h / output.h;
	int tx = threadIdx.x%sx;
	int ty = threadIdx.x/sx;
	int x = blockIdx.x%output.w; 
	int y = blockIdx.x/output.w;
	int m = blockIdx.y;

	int tid = threadIdx.x;
	__shared__ T smem[nthreads];
	smem[tid] = 0;

	if(tx < sx && ty < sy){
		smem[tid] = inputs(x*sx + tx, y*sy + ty, m);
	}
	volatile T* vsmem = smem;
	////Reduction
	__syncthreads();
	if(nthreads >= 256){
		if(tid < 128) vsmem[tid] += vsmem[tid+128];
		__syncthreads();
	}
	if(nthreads >= 128){
		if(tid < 64) 
			vsmem[tid] += vsmem[tid+64];
		__syncthreads();
	}

	//All these run in a single warp
	if(nthreads >= 64) if(tid < 32) vsmem[tid] = max(vsmem[tid + 32], vsmem[tid]);
	if(nthreads >= 32) if(tid < 16) vsmem[tid] = max(vsmem[tid + 16], vsmem[tid]);
	if(nthreads >= 16) if(tid < 8) vsmem[tid] = max(vsmem[tid + 8], vsmem[tid]);
	if(nthreads >= 8)  if(tid < 4) vsmem[tid] = max(vsmem[tid + 4], vsmem[tid]);
	if(nthreads >= 4)  if(tid < 2) vsmem[tid] = max(vsmem[tid + 2], vsmem[tid]);
	if(nthreads >= 2)  if(tid < 1) vsmem[tid] = max(vsmem[tid + 1], vsmem[tid]);

	if(tid == 0)
		output(x,y,m) = vsmem[0];
}


template <class T>
void PoolingLayer<TensorGPU, T>::Propagate(const TensorGPU<T>& layer_input )
{
	dim3 blocks(this->out().w()*this->out().h(),this->out().d(),1); 
	int nthreads = iRoundUpPow2(this->sx_*this->sy_);
	dim3 threads(nthreads,1,1);
    switch(this->pooling_type_)
    {
        //Without cast CUDA compiler gives a warning that expression must be an integral type
    case static_cast<int>(PoolingLayer<TensorGPU, T>::eAverage):
	    switch(nthreads)
	    {
	    case 1  : SubsampleKernel<T, 1 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 2  : SubsampleKernel<T, 2 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 4  : SubsampleKernel<T, 4 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 8  : SubsampleKernel<T, 8 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 16 : SubsampleKernel<T, 16 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 32 : SubsampleKernel<T, 32 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 64 : SubsampleKernel<T, 64 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 128: SubsampleKernel<T, 128 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 256: SubsampleKernel<T, 256 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    default:
		    throw std::runtime_error("Incorrect threads number in Propagate");
	    }
        break;
    case static_cast<int>(PoolingLayer<TensorGPU, T>::eMax):
	    switch(nthreads)
	    {
	    case 1  : MaxPoolingKernel<T, 1 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 2  : MaxPoolingKernel<T, 2 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 4  : MaxPoolingKernel<T, 4 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 8  : MaxPoolingKernel<T, 8 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 16 : MaxPoolingKernel<T, 16 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 32 : MaxPoolingKernel<T, 32 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 64 : MaxPoolingKernel<T, 64 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 128: MaxPoolingKernel<T, 128 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    case 256: MaxPoolingKernel<T, 256 ><<<blocks, threads>>>(layer_input,  
				     this->out_); break;
	    default:
		    throw std::runtime_error("Incorrect threads number in Propagate");
	    }
        break;
    default:
        throw std::runtime_error("Unknown pooling type");

    }
	cutilCheckMsg("Failed to propagate data in player on cuda");
}

template <class T, int nthreads, bool hessian>
__global__ void BakpropagateSubsampleKernel(TensorDev3<T> dedx, TensorDev3<T> de_dx_prev)
{
	int sx = de_dx_prev.w / dedx.w;
	int sy = de_dx_prev.h / dedx.h;
	int tx = threadIdx.x%sx;
	int ty = threadIdx.x/sx;
	int x = blockIdx.x%dedx.w; 
	int y = blockIdx.x/dedx.w;
	int m = blockIdx.y;

	if(tx < sx && ty < sy){
		de_dx_prev(x*sx + tx, y*sy + ty, m) = dedx(x,y,m)/(sx*sy);
	}
}
template <class T, int nthreads, bool hessian>
__global__ void BakpropagateMaxPoolingKernel(TensorDev3<T> input, TensorDev3<T> output,
                                             TensorDev3<T> dedx, TensorDev3<T> de_dx_prev)
{
	int sx = de_dx_prev.w / dedx.w;
	int sy = de_dx_prev.h / dedx.h;
	int tx = threadIdx.x%sx;
	int ty = threadIdx.x/sx;
	int x = blockIdx.x%dedx.w; 
	int y = blockIdx.x/dedx.w;
	int m = blockIdx.y;

	if(tx < sx && ty < sy){
		de_dx_prev(x*sx + tx, y*sy + ty, m) = 
            input(x*sx + tx, y*sy + ty, m) == output(x,y,m) ? dedx(x,y,m) : 0;
	}
}


template <class T>
template <bool hessian>
void PoolingLayer<TensorGPU, T>::BackpropagateKernelProxy(const TensorGPU<T>& input, const TensorGPU<T>& dedx_prev)
{
	const TensorGPU<T>& de_dx_in = hessian ? this->d2e_dx2() : this->de_dx();
	dim3 blocks(this->out().w()*this->out().h(),this->out().d(),1); 
	int nthreads = iRoundUpPow2(this->sx_*this->sy_);
	dim3 threads(nthreads,1,1);
    switch(this->pooling_type_)
    {
    case static_cast<int>(PoolingLayer<TensorGPU, T>::eAverage):
	    switch(nthreads)
	    {
	    case 1  : BakpropagateSubsampleKernel<T, 1 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 2  : BakpropagateSubsampleKernel<T, 2 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 4  : BakpropagateSubsampleKernel<T, 4 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 8  : BakpropagateSubsampleKernel<T, 8 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 16 : BakpropagateSubsampleKernel<T, 16 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 32 : BakpropagateSubsampleKernel<T, 32 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 64 : BakpropagateSubsampleKernel<T, 64 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 128: BakpropagateSubsampleKernel<T, 128 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    case 256: BakpropagateSubsampleKernel<T, 256 , hessian><<<blocks, threads>>>(de_dx_in,  
				      dedx_prev); break;
	    default:
		    throw std::runtime_error("Incorrect threads number in Propagate");
	    }
        break;
    case static_cast<int>(PoolingLayer<TensorGPU, T>::eMax):
	    switch(nthreads)
	    {
	    case 1  : BakpropagateMaxPoolingKernel<T, 1 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 2  : BakpropagateMaxPoolingKernel<T, 2 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 4  : BakpropagateMaxPoolingKernel<T, 4 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 8  : BakpropagateMaxPoolingKernel<T, 8 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 16 : BakpropagateMaxPoolingKernel<T, 16 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 32 : BakpropagateMaxPoolingKernel<T, 32 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 64 : BakpropagateMaxPoolingKernel<T, 64 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 128: BakpropagateMaxPoolingKernel<T, 128 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    case 256: BakpropagateMaxPoolingKernel<T, 256 , hessian><<<blocks, threads>>>(input, this->out_, de_dx_in,  
				      dedx_prev); break;
	    default:
		    throw std::runtime_error("Incorrect threads number in Propagate");
	    }
        break;
    default:
        throw std::runtime_error("Unknown pooling type");

    }
}


template <class T>
void PoolingLayer<TensorGPU, T>::BackPropagate(const TensorGPU<T>& input, TensorGPU<T>& dedx_prev)
{
	assert(dedx_prev.HaveSameSize(input));
	BackpropagateKernelProxy<false>(input, dedx_prev);
}

template <class T>
void PoolingLayer<TensorGPU, T>::BackPropagateHessian(const TensorGPU<T>& input, TensorGPU<T>& d2edx2_prev)
{
	assert(d2edx2_prev.HaveSameSize(input));
	BackpropagateKernelProxy<true>(input, d2edx2_prev);
}
#endif //HAVE_CUDA
}
